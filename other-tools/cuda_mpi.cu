#include "hip/hip_runtime.h"
#include "mpi.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
__global__ void touch_data(char*A,size_t size) {
   for(size_t i=0;i<size;i++) {
      A[i] = 42;
   }
}
void bw_test(int min_msg_size, int max_msg_size, int global_rank, int global_size)
{
    int window_size = 10;
    double t_start, t_end, t_total;
    int iterations = 10;
    int skips = 2;
    char *gpu_src_buf, *gpu_dst_buf;
    MPI_Request request_s[global_size];
    MPI_Request request_r[global_size];
    MPI_Status reqstat[global_size];
    MPI_Status stat;
//    if (global_rank == src || global_rank == dst)
//    {
        gpuErrchk(hipMalloc(&gpu_src_buf, max_msg_size*global_size));
        gpuErrchk(hipMalloc(&gpu_dst_buf, max_msg_size*global_size));
    touch_data<<<1,1>>>(gpu_src_buf, max_msg_size*global_size);
    gpuErrchk(hipDeviceSynchronize());
//    }
    for(int size = min_msg_size; size <= max_msg_size; size *= 2)
    {
        MPI_Barrier(MPI_COMM_WORLD);
        t_total = 0.0;
        for(int i = 0; i < iterations + skips; ++i)
        {
           if(i >= skips) {
              t_start = MPI_Wtime();
           }
           for(int dst=0;dst<global_size;dst++)
            {
//               printf("Sending %d to %d\n",i,dst);
//                for(int j = 0; j < window_size; j++) {
                    MPI_Isend(&gpu_src_buf[size*dst], size, MPI_CHAR, dst, 100, MPI_COMM_WORLD, &request_s[dst]);
//                }
                MPI_Irecv(&gpu_dst_buf[size*dst], size, MPI_CHAR, dst, 100, MPI_COMM_WORLD, &request_r[dst]);
            }
           MPI_Waitall(global_size, request_r, MPI_STATUSES_IGNORE);
           MPI_Waitall(global_size, request_s, MPI_STATUSES_IGNORE);
           if (i >= skips) {
              t_end = MPI_Wtime();
              t_total += (t_end - t_start);
           }
        }
        MPI_Barrier(MPI_COMM_WORLD);
//        if (global_rank == src) {
            double tmp = size / 1e6 * iterations * window_size;
            double bw = tmp / t_total;
            printf( "size=%d bw=%f MB/s\n", size, bw);
//            fflush(stdout);
//        }
    }
        hipFree(gpu_src_buf);
        hipFree(gpu_dst_buf);
}
int main(int argc, char** argv)
{
    int global_rank, global_size;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &global_rank);
    MPI_Comm_size(MPI_COMM_WORLD, &global_size);
    hipSetDevice(global_rank);
    int min_msg_size = 8192;
    int max_msg_size = 2097152;
//    if (global_rank == 0)
//        printf("src %d --> dst %d \n", src, dst);
//    int src=0, dst=1;
    bw_test(min_msg_size, max_msg_size, global_rank, global_size);
    MPI_Barrier(MPI_COMM_WORLD);
//    src=0, dst=2;
//    if (global_rank == 0)
//        printf("src %d --> dst %d \n", src, dst);
    bw_test(min_msg_size, max_msg_size, global_rank, global_size);
    MPI_Finalize();
    return 0;
}
